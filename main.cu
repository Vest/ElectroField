#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

/* screen constants */
const int width = 1024;
const int height = 768;

/* charge constants */
const float k = 20.0f;
const float minDistance = 0.1f; // not to divide by zero
const float maxSolidColorLength = 1.0f;

/* charges on the field */
const int maxCharge = 1000;
const int minCharge = -1000;

const char maxChargeCount = 30;
char chargeCount = 0;
__constant__ char dev_chargeCount;

float3 charges[maxChargeCount]; // x, y, z == m
__constant__ float3 dev_charges[maxChargeCount]; // x, y, z == m

/* OpenGL interoperability */
dim3 blocks, threads;

GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;

/* charge selection */
const int detectChargeRange = 20;
int selectedChargeIndex = -1;
bool isDragging = false;

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void key(unsigned char key, int x, int y) {
	switch (key) {
	case 27:
		printf("Exit application\n");

		glutLeaveMainLoop();
		break;
	}
}

__device__ float length(const float2& q) {
	return sqrtf(q.x * q.x + q.y * q.y);
}

__device__ float length2(const float2& q) {
	return (q.x * q.x + q.y * q.y);
}

__device__ void setColor(const float2& f, uchar4& pixel) {
	pixel.x = pixel.y = pixel.z = pixel.w = 0;

	float l = length(f);
	pixel.x = (l > maxSolidColorLength ? 255 : l * 256 / maxSolidColorLength);
}

__device__ void calculate(const float3& charge, int x, int y, float2& f) {
	f.x = x - charge.x;
	f.y = y - charge.y;

	float l = length2(f) + minDistance;

	float e = charge.z * rsqrt(l * l * l);
	f.x *= e;
	f.y *= e;
}

__global__ void renderFrame(uchar4* screen) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float2 force, t_force;
	force.x = force.y = 0.0f;

	if (x >= width || y >= height)
		return;

	for (char i = 0; i < dev_chargeCount; i++) {
		calculate(dev_charges[i], x, y, t_force);

		force.x += t_force.x;
		force.y += t_force.y;
	}

	force.x *= k;
	force.y *= k;

	setColor(force, screen[x + y * width]);
}

void idle(void) {
	uchar4* dev_screen;
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
			hipGraphicsResourceGetMappedPointer((void**) &dev_screen, &size, cuda_vbo_resource));

	// Kernel Time measure
	hipEvent_t startEvent, stopEvent;
	float elapsedTime = 0.0f;
	HANDLE_ERROR(hipEventCreate(&startEvent));
	HANDLE_ERROR(hipEventCreate(&stopEvent));
	HANDLE_ERROR(hipEventRecord(startEvent, 0));

	// Render Image
	renderFrame<<<blocks, threads>>>(dev_screen);
	HANDLE_ERROR(hipDeviceSynchronize());

	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

	// Kernel Time measure
	HANDLE_ERROR(hipEventRecord(stopEvent, 0));
	HANDLE_ERROR(hipEventSynchronize(stopEvent));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));

	char fps[256];
	sprintf(fps, "Electric field: %3.4f ms per frame (FPS: %3.1f)", elapsedTime,
			1000 / elapsedTime);
	glutSetWindowTitle(fps);

	glutPostRedisplay();
}

void draw(void) {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);

	glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glPointSize(3.0f);
	glColor3f(0.0f, 1.0f, 1.0f);
	glBegin(GL_POINTS);
	glVertex2i(charges[selectedChargeIndex].x, charges[selectedChargeIndex].y);
	glEnd();

	glutSwapBuffers();
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
		unsigned int vbo_res_flags) {
	unsigned int size = width * height * sizeof(uchar4);

	glGenBuffers(1, vbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, *vbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, size, NULL, GL_DYNAMIC_DRAW);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res) {
	HANDLE_ERROR(hipGraphicsUnregisterResource(cuda_vbo_resource));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

void pushCharge(int x, int y) {
	if (chargeCount < maxChargeCount)
		chargeCount++;
	else {
		for (int i = 0; i < maxChargeCount - 1; ++i) {
			charges[i] = charges[i + 1];
		}
	}

	charges[chargeCount - 1].x = x;
	charges[chargeCount - 1].y = y;
	charges[chargeCount - 1].z = rand() % (maxCharge - minCharge) + minCharge;

	printf("Debug: Charge #%d (%.0f, %.0f, %.0f)\n", chargeCount - 1,
			charges[chargeCount - 1].x, charges[chargeCount - 1].y,
			charges[chargeCount - 1].z);

	HANDLE_ERROR(
			hipMemcpyToSymbol(HIP_SYMBOL(dev_charges), charges, chargeCount * sizeof(float3)));
	HANDLE_ERROR(
			hipMemcpyToSymbol(HIP_SYMBOL(dev_chargeCount), &chargeCount, sizeof(chargeCount)));
	printf("Charges %d\n", chargeCount);
}

void mouse(int button, int state, int x, int y) {
	if (button != GLUT_LEFT_BUTTON)
		return;

	if (state == GLUT_DOWN) {
		if (selectedChargeIndex != -1) { // Drag
			printf("Drag charge #%d... ", selectedChargeIndex);
			isDragging = true;
		}
	} else {
		if (selectedChargeIndex != -1) { // Drop
			printf("Drop\n");
			isDragging = false;
		} else {
			pushCharge(x, height - y);
		}
	}
}

void mouseDrag(int x, int y) {
	if (isDragging && selectedChargeIndex != -1) {
		printf(" drag... ");
		charges[selectedChargeIndex].x = x;
		charges[selectedChargeIndex].y = height - y;

		HANDLE_ERROR(
				hipMemcpyToSymbol(HIP_SYMBOL(dev_charges), charges, chargeCount * sizeof(float3)));
	}
}

void mouseTrack(int x, int y) {
	if (isDragging)
		return;
	// Detect selected charge
	int dx = 0, dy = 0;

	for (int i = 0; i < chargeCount; i++) {
		dx = x - charges[i].x;
		dy = (height - y) - charges[i].y;

		if (dx * dx + dy * dy < detectChargeRange * detectChargeRange) {
			selectedChargeIndex = i;

			return;
		}
	}

	selectedChargeIndex = -1;
}

void initCuda(int deviceId) {
	int deviceCount = 0;
	HANDLE_ERROR(hipGetDeviceCount(&deviceCount));

	if (deviceCount <= 0) {
		printf("No CUDA devices found\n");
		exit(-1);
	}

	HANDLE_ERROR(cudaGLSetGLDevice(deviceId));

	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties, deviceId));

	threads.x = 32;
	threads.y = properties.maxThreadsPerBlock / threads.x - 2; // to avoid hipErrorLaunchOutOfResources error

	blocks.x = (width + threads.x - 1) / threads.x;
	blocks.y = (height + threads.y - 1) / threads.y;

	printf(
			"Debug: blocks(%d, %d), threads(%d, %d)\nCalculated Resolution: %d x %d\n",
			blocks.x, blocks.y, threads.x, threads.y, blocks.x * threads.x,
			blocks.y * threads.y);
}

void initGlut(int argc, char** argv) {
	// Initialize freeglut
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(width, height);
	glutCreateWindow("Electric field");
	glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);

	glutIdleFunc(idle);
	glutKeyboardFunc(key);
	glutMouseFunc(mouse);
	glutMotionFunc(mouseDrag);
	glutPassiveMotionFunc(mouseTrack);
	glutDisplayFunc(draw);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble) width, 0.0, (GLdouble) height);

	glewInit();
}

int main(int argc, char** argv) {
	setbuf(stdout, NULL);

	initCuda(0);
	initGlut(argc, argv);

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	glutMainLoop();

	deleteVBO(&vbo, cuda_vbo_resource);

	return 0;
}
